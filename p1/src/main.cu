
#include <hip/hip_runtime.h>
#include <cstdio>

// CUDA kernel — выполняется на GPU
__global__ void cuda_hello() { printf("Hello World from GPU1!\n"); }

int main() {
    // Запуск kernel: 1 блок, 1 поток
    cuda_hello<<<1, 1>>>();

    // Ждём завершения всех потоков GPU перед завершением программы
    hipDeviceSynchronize();

    // Можно проверить, была ли ошибка запуска kernel
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
        return 1;
    }

    printf("Hello World from CPU1!\n");
    return 0;
}